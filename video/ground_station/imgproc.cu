#include "hip/hip_runtime.h"
#include "highgui.h"
#include "imgproc.h"
#include <cmath>
#include "gpu_util.h"
#include <cassert>
#include <cstdio>

#define M_PI_8   0.39269908169872414
#define M_3_PI_8 1.1780972450961724
#define M_5_PI_8 1.9634954084936207
#define M_7_PI_8 2.748893571891069

__global__ void GPU_bilinear_filter2(float *G, float *F, int G_rows, int G_cols, int F_rows, int F_cols, int n_slices, float kr, float kc) {
  // gather
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= G_rows || j >= G_cols) {
    return;
  }
  float y = (float)i * kr - 0.5f;
  float x = (float)j * kc - 0.5f;
  int Fi = (int)floorf(y);
  int Fj = (int)floorf(x);
  float dy = 1.0f - (y - floorf(y));
  float dx = 1.0f - (x - floorf(x));
  float wsum = 0.0f;
  float total = 0.0f;
  for (int k = 0; k < n_slices; k++) {
    if (Fj >= 0 && Fj < F_cols) {
      if (Fi >= 0 && Fi < F_rows) {
        wsum += dx * dy * F[IJK2C(Fi, Fj, k, F_rows, F_cols)];
        total += dx * dy;
      }
      if (Fi+1 >= 0 && Fi+1 < F_rows) {
        wsum += dx * (1-dy) * F[IJK2C(Fi+1, Fj, k, F_rows, F_cols)];
        total += dx * (1-dy);
      }
    }
    if (Fj+1 >= 0 && Fj+1 < F_cols) {
      if (Fi >= 0 && Fi < F_rows) {
        wsum += (1-dx) * dy * F[IJK2C(Fi, Fj+1, k, F_rows, F_cols)];
        total += (1-dx) * dy;
      }
      if (Fi+1 >= 0 && Fi < F_cols) {
        wsum += (1-dx) * (1-dy) * F[IJK2C(Fi+1, Fj+1, k, F_rows, F_cols)];
        total += (1-dx) * (1-dy);
      }
    }
    if (total != 0.0f) {
      G[IJK2C(i, j, k, G_rows, G_cols)] = wsum / total; // normalize
    }
  }
}

gcube gpu_imresize2(const gcube &A, int m, int n) {
  gcube G(m, n, A.n_slices);
  double kr = (double)A.n_rows / (double)m;
  double kc = (double)A.n_cols / (double)n;
  dim3 blockSize(16, 16, 1);
  dim3 gridSize((G.n_cols-1)/16+1, (G.n_rows-1)/16+1, 1);
  GPU_bilinear_filter2<<<gridSize, blockSize>>>(G.d_pixels, A.d_pixels,
      G.n_rows, G.n_cols, A.n_rows, A.n_cols, A.n_slices, (float)kr, (float)kc);
  checkCudaErrors(hipGetLastError());
  return G;
}
